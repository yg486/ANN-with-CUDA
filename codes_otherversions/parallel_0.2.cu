#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <math.h>

#include <iostream>  
#include <string>  
#include <vector>  
#include <fstream>  
#include <sstream> 
#include<random> 
using namespace std; 


#define X_trn(x, y) X_trn[x * size_train + y] // 196 * 964
#define X_tst(x, y) X_tst[x * size_test + y]  // 196 * 414
#define Y_trn(x, y) Y_trn[x * size_train + y] // 1   * 964
#define Y_tst(x, y) Y_tst[x * size_test + y]  // 1   * 414
#define X(x, y) X[x * size_batch + y]  // 196 * 964
#define Y(x, y) Y[x * size_batch + y]  // 1   * 414


#define W1(x, y) W1[x * size_input + y]       // 20 * 196
#define b1(x, y) b1[x * 1 + y]                // 20 * 1
#define W2(x, y) W2[x * size_hidden + y]      // 2  * 20
#define b2(x, y) b2[x * 1 + y]                // 2  * 1

#define dW1(x, y) dW1[x * size_input + y]     // 20 * 196
#define db1(x, y) db1[x * 1 + y]              // 20 * 1
#define dW2(x, y) dW2[x * size_hidden + y]    // 2  * 20
#define db2(x, y) db2[x * 1 + y]              // 2  * 1

#define Z1(x, y) Z1[x * size_batch + y]       // 20 * 964
#define A1(x, y) A1[x * size_batch + y]       // 20 * 964
#define Z2(x, y) Z2[x * size_batch + y]       // 2  * 964
#define A2(x, y) A2[x * size_batch + y]       // 2  * 964

#define dZ1(x, y) dZ1[x * size_batch + y]     // 20 * 964
#define dA1(x, y) dA1[x * size_batch + y]     // 20 * 964
#define dZ2(x, y) dZ2[x * size_batch + y]     // 2  * 964
#define dA2(x, y) dA2[x * size_batch + y]     // 2  * 964


#define dev_X_trn(x, y) dev_X_trn[x * size_train + y] // 196 * 964
#define dev_X_tst(x, y) dev_X_tst[x * size_test + y]  // 196 * 414
#define dev_Y_trn(x, y) dev_Y_trn[x * size_train + y] // 1   * 964
#define dev_Y_tst(x, y) dev_Y_tst[x * size_test + y]  // 1   * 414
#define dev_X(x, y) dev_X[x * size_batch + y] // 196 * 964
#define dev_Y(x, y) dev_Y[x * size_batch + y]  // 1   * 414


#define dev_W1(x, y) dev_W1[x * size_input + y]       // 20 * 196
#define dev_b1(x, y) dev_b1[x * 1 + y]                // 20 * 1
#define dev_W2(x, y) dev_W2[x * size_hidden + y]      // 2  * 20
#define dev_b2(x, y) dev_b2[x * 1 + y]                // 2  * 1

#define dev_dW1(x, y) dev_dW1[x * size_input + y]     // 20 * 196
#define dev_db1(x, y) dev_db1[x * 1 + y]              // 20 * 1
#define dev_dW2(x, y) dev_dW2[x * size_hidden + y]    // 2  * 20
#define dev_db2(x, y) dev_db2[x * 1 + y]              // 2  * 1

#define dev_Z1(x, y) dev_Z1[x * size_batch + y]       // 20 * 964
#define dev_A1(x, y) dev_A1[x * size_batch + y]       // 20 * 964
#define dev_Z2(x, y) dev_Z2[x * size_batch + y]       // 2  * 964
#define dev_A2(x, y) dev_A2[x * size_batch + y]       // 2  * 964

#define dev_dZ1(x, y) dev_dZ1[x * size_batch + y]     // 20 * 964
#define dev_dA1(x, y) dev_dA1[x * size_batch + y]     // 20 * 964
#define dev_dZ2(x, y) dev_dZ2[x * size_batch + y]     // 2  * 964
#define dev_dA2(x, y) dev_dA2[x * size_batch + y]     // 2  * 964

#define max_index(x, y) max_index[y] // 1  * 964

int size_train  = 964;
int size_test   = 414;
int size_batch  = 0;

int size_input  = 196;
int size_hidden = 20;
int size_output = 2;

int size_X_trn = 196*964;
int size_Y_trn = 1*964;
int size_X_tst = 196*414;
int size_Y_tst = 1*414;
int size_Xbatch = 0;
int size_Ybatch = 0;


int size_W1 = size_hidden*size_input;
int size_b1 = size_hidden*1;
int size_W2 = size_output*size_hidden;
int size_b2 = size_output*1;

int size_dW1 = size_hidden*size_input;
int size_db1 = size_hidden*1;
int size_dW2 = size_output*size_hidden;
int size_db2 = size_output*1;

#define size_Z1 size_hidden*size_batch
#define size_A1 size_hidden*size_batch
#define size_Z2 size_output*size_batch
#define size_A2 size_output*size_batch

#define size_dZ1 size_hidden*size_batch
#define size_dA1 size_hidden*size_batch
#define size_dZ2 size_output*size_batch
#define size_dA2 size_output*size_batch

#define size_max_index 1*size_batch

double *X_trn, *X_tst;
int *Y_trn, *Y_tst;
double *W1, *b1, *W2, *b2;
double *dW1, *db1, *dW2, *db2;
double *Z1, *A1, *Z2, *A2;
double *dZ1, *dA1, *dZ2, *dA2;
int *max_index;



__global__ void HiddenLayer(double* dev_X, double* dev_W1, double* dev_b1, double* dev_A1, double* dev_Z1, int size_input, int size_batch, int acti_type)

{

	int k;
	int i = blockIdx.x; // row of A1
	int j = threadIdx.x; // column of A1
	double partial = 0.0;
 
	for (k = 0; k < size_input; k++)
		partial += dev_W1(i,k) * dev_X(k,j);
	dev_Z1(i,j) = partial + dev_b1(i,0);
 
	// Sigmoid
	if (acti_type == 1)
		dev_A1(i,j) = 1 / (1 + exp(0 - dev_Z1(i,j)));
  
	// ReLU
	if (acti_type == 2) {
		if (dev_Z1(i,j) < 0)
			dev_A1(i,j) = 0;
		if (dev_Z1(i,j) >= 0)
			dev_A1(i,j) = dev_Z1(i,j);
	}
  
}

__global__ void OutputLayer(double* dev_A1, double* dev_W2, double* dev_b2, double* dev_Z2, int size_hidden, int size_batch)

{

	int k;
	int i = blockIdx.x; // row of Z2
	int j = threadIdx.x; // column of Z2
	double partial = 0.0;
 
	for (k = 0; k < size_hidden; k++)
		partial += dev_W2(i,k) * dev_A1(k,j);
	dev_Z2(i,j) = partial + dev_b2(i,0);
 
}

void Softmax(double* Z2, int row, int col, double* A2, int* max_index)
{

  int c, r;  
	double max = 0, sum = 0;
	for (c = 0; c < col; c++) {
    max = Z2(0, c);
    max_index[c] = 1;    
		for (r = 1; r < row; r++) {   
			if (Z2(r, c) > max){      
				max = Z2(r, c);        
        max_index[c] = 0;        
      }
		}
		sum = 0;
		for (r = 0; r < row; r++)
			sum += exp(Z2(r, c));
		for (r = 0; r < row; r++)
			A2(r, c) = exp(Z2(r, c)) / sum;
  }
  return;

}


double cross_entropy_loss(int* Y, double* A2, int col) 
{
  
  int c;
  double loss = 0;
  for(c = 0; c < col; c++) {
    loss += -log(A2(0, c)) * Y(0, c) - log(A2(1, c)) * (1-Y(0, c));
  }
  return loss/col;
  
}

/* init Z and A in the host */
void initialize_ZA(int size_batch) {

  Z1 = (double *) malloc(size_Z1*sizeof(double));   // 20*964
  A1 = (double *) malloc(size_A1*sizeof(double));   // 20*964
  Z2 = (double *) malloc(size_Z2*sizeof(double));   // 2*964
  A2 = (double *) malloc(size_A2*sizeof(double));   // 2*964

  dZ1 = (double *) malloc(size_dZ1*sizeof(double));  // 20*964
  dA1 = (double *) malloc(size_dA1*sizeof(double));  // 20*964
  dZ2 = (double *) malloc(size_dZ2*sizeof(double));  // 2*964
  dA2 = (double *) malloc(size_dA2*sizeof(double));  // 2*964
  
  max_index = (int *) malloc(size_max_index*sizeof(int));             // 1*964
    
  memset (Z1,0,  size_Z1);
  memset (A1,0,  size_A1);
  memset (Z2,0,  size_Z2);
  memset (A2,0,  size_A2);
  
  memset (dZ1,0, size_dZ1);
  memset (dA1,0, size_dA1);
  memset (dZ2,0, size_dZ2);
  memset (dA2,0, size_dA2);
  
  memset (max_index,0,size_max_index);

}

void forward(double* X, int* Y, string type, int acti_type){

  if(type == "train"){
    size_batch  = size_train;
    size_Xbatch = size_X_trn;
    size_Ybatch = size_Y_trn;        
  }
  else{
    size_batch = size_test;
    size_Xbatch = size_X_tst;
    size_Ybatch = size_Y_tst;    
  }

  // init Z and A in the host
  initialize_ZA(size_batch);

  // init X Y W b Z A in the device
  double *dev_X, *dev_W1, *dev_b1, *dev_W2, *dev_b2, *dev_Z1, *dev_A1, *dev_Z2, *dev_A2;
  int *dev_Y;
  
  hipMalloc((void**)&dev_X,  size_Xbatch *  sizeof(double));
  hipMalloc((void**)&dev_Y,  size_Ybatch *  sizeof(int));
  
  hipMalloc((void**)&dev_W1, size_W1 * sizeof(double));
  hipMalloc((void**)&dev_b1, size_b1 * sizeof(double));
  hipMalloc((void**)&dev_W2, size_W2 * sizeof(double));
  hipMalloc((void**)&dev_b2, size_b2 * sizeof(double));

  hipMalloc((void**)&dev_Z1, size_Z1 * sizeof(double));
  hipMalloc((void**)&dev_A1, size_A1 * sizeof(double));  
  hipMalloc((void**)&dev_Z2, size_Z2 * sizeof(double));
  hipMalloc((void**)&dev_A2, size_A2 * sizeof(double)); 

  // hidden layer and activation function to get Z1 and A1
  hipMemcpy(dev_W1, W1, size_W1 * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_b1, b1, size_b1 * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_X,  X,  size_Xbatch  * sizeof(double), hipMemcpyHostToDevice);
  
  HiddenLayer<<<size_hidden, size_batch>>>(dev_X, dev_W1, dev_b1, dev_A1, dev_Z1, size_input, size_batch, acti_type); // 1 is Sigmoid

  hipMemcpy(Z1, dev_Z1, size_Z1 * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(A1, dev_A1, size_A1 * sizeof(double), hipMemcpyDeviceToHost);

  /*
  printf("**************\n");
  for (int p = 0; p < 5; p++) {
    for(int q = 0; q < 6; q++) {
      printf("%f ", Z1(p, q));
    }
    printf("\n");
  }
  printf("**************\n");
  for (int p = 0; p < 5; p++) {
    for(int q = 0; q < 6; q++) {
      printf("%f ", A1(p, q));
    }
    printf("\n");
  }
  */
 
  // output layer to get Z2
  hipMemcpy(dev_W2, W2, size_W2 * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_b2, b2, size_b2 * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_A1, A1, size_A1 * sizeof(double), hipMemcpyHostToDevice);
  
  OutputLayer<<<size_output, size_batch>>>(dev_A1, dev_W2, dev_b2, dev_Z2, size_hidden, size_batch);

  hipMemcpy(Z2, dev_Z2, size_Z2 * sizeof(double), hipMemcpyDeviceToHost);

  /*
  printf("**************\n");
  for (int p = 0; p < 5; p++) {
    for(int q = 0; q < 6; q++) {
      printf("%f ", Z2(p, q));
    }
    printf("\n");
  }  
  */
  
  // softmax layer to get A2

  Softmax(Z2, size_output, size_batch, A2, max_index);
  
  /*
  printf("**************\n");
  for (int p = 0; p < 1; p++) {
    for(int q = 0; q < 6; q++) {
      printf("%d ", max_index(p, q));
    }
    printf("\n");
  }
  */

  hipFree(dev_X);
  hipFree(dev_Y);   
  hipFree(dev_W1);
  hipFree(dev_b1); 
  hipFree(dev_W2);
  hipFree(dev_b2);
  hipFree(dev_Z1);
  hipFree(dev_A1);  
  hipFree(dev_Z2);
  hipFree(dev_A2);

}


__global__ void Back_dZ2 (double* dev_A2, int* dev_Y_trn, double* dev_dZ2, int size_train, int size_batch)

{

	int c = threadIdx.x; // column of Z2
  dev_dZ2(0, c) = (dev_A2(0, c) - dev_Y_trn(0, c)) / size_train;
  dev_dZ2(1, c) = (dev_Y_trn(0, c) - dev_A2(0, c)) / size_train;

}

// dW1(20*196) = dZ1(20*964) * X(196*964)
// dW2(2*20) = dZ2(2*964) * A1(20*964)
__global__ void Back_dW (double* dev_A, double* dev_dZ, double* dev_dW, int size_batch, int W_col)

{  

	int k;
	int i = blockIdx.x; // row of Z2
	int j = threadIdx.x; // column of Z2
	double tmp = 0.0;
 
	for (k = 0; k < size_batch; k++)
		tmp += dev_dZ[i*size_batch+k] * dev_A[j*size_batch+k];
	dev_dW[i*W_col+j] = tmp;

}

// db1(20*1) is from dZ1(20*964)
// db2(2*1) is from dZ1(2*964)
void Back_db(double* dZ, double* db, int row, int col, int size_batch)

{
  int r, c;
  for(r = 0; r < row; r++) {
    double tmp = 0;
    for(c = 0; c < col; c++) {
      tmp += dZ[r*size_batch+c];
    }
    db[r*1+0] = tmp;
  }
}
    
__global__ void Back_dA1 (double* dev_W2, double* dev_dZ2, double* dev_dA1, int size_batch, int size_hidden, int size_output)

{  
    
  // dA1(20*964) = dZ2(2*964) * W2(2*20)
	int k;
	int i = blockIdx.x; // 20
	int j = threadIdx.x; // 964
	double partial = 0.0;
 
	for (k = 0; k < size_output; k++)
		partial += dev_W2(k,i) * dev_dZ2(k,j);
	dev_dA1(i,j) = partial;

}


__global__ void Back_dZ1 (double* dev_dA1, double* dev_A1, double* dev_Z1, double* dev_dZ1, int size_batch, int acti_type)

{  

	int i = blockIdx.x; // 20
	int j = threadIdx.x; // 964

  if(acti_type == 1){ // Sigmoid
      dev_dZ1(i, j) = dev_dA1(i, j) * dev_A1(i, j) * (1-dev_A1(i, j)); // dZ1 = dA1*A1*(1-A1)
  } 
  else if(acti_type == 2) { // ReLU
    if(dev_Z1(i, j) < 0) 
      dev_dZ1(i, j) = 0;
    else
      dev_dZ1(i, j) = dev_dA1(i, j); //dZ1 = dA1*Z1_mask
  }

}




void backprop(int acti_type) { // type = 1 is Sigmoid

  double *dev_X_trn, *dev_W1, *dev_b1, *dev_W2, *dev_b2, *dev_Z1, *dev_A1, *dev_Z2, *dev_A2;
  double *dev_dW1, *dev_db1, *dev_dW2, *dev_db2, *dev_dZ1, *dev_dA1, *dev_dZ2, *dev_dA2;
  int *dev_Y_trn;
  
  hipMalloc((void**)&dev_X_trn,  size_X_trn *  sizeof(double));
  hipMalloc((void**)&dev_Y_trn,  size_Y_trn *  sizeof(int));
  
  hipMalloc((void**)&dev_W1, size_W1 * sizeof(double));
  hipMalloc((void**)&dev_b1, size_b1 * sizeof(double));
  hipMalloc((void**)&dev_W2, size_W2 * sizeof(double));
  hipMalloc((void**)&dev_b2, size_b2 * sizeof(double));

  hipMalloc((void**)&dev_Z1, size_Z1 * sizeof(double));
  hipMalloc((void**)&dev_A1, size_A1 * sizeof(double));  
  hipMalloc((void**)&dev_Z2, size_Z2 * sizeof(double));
  hipMalloc((void**)&dev_A2, size_A2 * sizeof(double)); 
  
  hipMalloc((void**)&dev_dW1, size_dW1 * sizeof(double));
  hipMalloc((void**)&dev_db1, size_db1 * sizeof(double));
  hipMalloc((void**)&dev_dW2, size_dW2 * sizeof(double));
  hipMalloc((void**)&dev_db2, size_db2 * sizeof(double));
  
  hipMalloc((void**)&dev_dZ1, size_dZ1 * sizeof(double));
  hipMalloc((void**)&dev_dA1, size_dA1 * sizeof(double));  
  hipMalloc((void**)&dev_dZ2, size_dZ2 * sizeof(double));
  hipMalloc((void**)&dev_dA2, size_dA2 * sizeof(double));

  // get dZ2
  hipMemcpy(dev_A2, A2, size_A2 * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_Y_trn, Y_trn, size_Y_trn * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_dZ2,  dZ2,  size_dZ2  * sizeof(double), hipMemcpyHostToDevice);

  Back_dZ2<<<1, size_train>>>(dev_A2, dev_Y_trn, dev_dZ2, size_train, size_train);

  hipMemcpy(dZ2, dev_dZ2, size_dZ2 * sizeof(double), hipMemcpyDeviceToHost);

  // get dw2
  hipMemcpy(dev_A1, A1, size_A1 * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_dZ2,  dZ2, size_dZ2  * sizeof(double), hipMemcpyHostToDevice);
  
  Back_dW<<<size_output, size_hidden>>>(dev_A1, dev_dZ2, dev_dW2, size_train, size_hidden);
  
  hipMemcpy(dW2, dev_dW2, size_dW2 * sizeof(double), hipMemcpyDeviceToHost);

  // get db2
  Back_db(dZ2, db2, size_output, size_train, size_train);

  // get dA1
  hipMemcpy(dev_W2, W2, size_W2 * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_dZ2, dZ2, size_dZ2 * sizeof(double), hipMemcpyHostToDevice);  
  
  Back_dA1<<<size_hidden, size_train>>> (dev_W2, dev_dZ2, dev_dA1, size_train, size_hidden, size_output);    
        
  hipMemcpy(dA1, dev_dA1, size_dA1 * sizeof(double), hipMemcpyDeviceToHost);

  // get dZ1
  hipMemcpy(dev_dA1, dA1, size_dA1 * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_A1, A1, size_A1 * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_Z1, Z1, size_Z1 * sizeof(double), hipMemcpyHostToDevice);
  
  Back_dZ1<<<size_hidden, size_train>>>(dev_dA1, dev_A1, dev_Z1, dev_dZ1, size_train, acti_type);

  hipMemcpy(dZ1, dev_dZ1, size_dZ1 * sizeof(double), hipMemcpyDeviceToHost);

  // get dW1

  hipMemcpy(dev_X_trn, X_trn, size_X_trn * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_dZ1,  dZ1, size_dZ1  * sizeof(double), hipMemcpyHostToDevice);
  
  Back_dW<<<size_hidden, size_input>>>(dev_X_trn, dev_dZ1, dev_dW1, size_train, size_input);

  hipMemcpy(dW1, dev_dW1, size_dW1 * sizeof(double), hipMemcpyDeviceToHost);

  // get b1
  Back_db(dZ1, db1, size_hidden, size_train, size_train);
  
 
  hipFree(dev_X_trn);
  hipFree(dev_Y_trn);
  hipFree(dev_W1);
  hipFree(dev_b1);
  hipFree(dev_W2);
  hipFree(dev_b2);
  hipFree(dev_Z1);
  hipFree(dev_A1);
  hipFree(dev_Z2);
  hipFree(dev_A2);
  hipFree(dev_dW1);
  hipFree(dev_db1);
  hipFree(dev_dW2);
  hipFree(dev_db2);
  hipFree(dev_dZ1);
  hipFree(dev_dA1);
  hipFree(dev_dZ2);
  hipFree(dev_dA2);

}

__global__ void update_Wb(double* dev_dWb, double* dev_Wb, int col, double learn_rate)
{

	int r = blockIdx.x; // row of Z2
	int c = threadIdx.x; // column of Z2
  
  dev_Wb[r*col+c] = dev_Wb[r*col+c] - learn_rate * dev_dWb[r*col+c];
  
}

void updateParameter(double learn_rate)
{

  double *dev_W1, *dev_b1, *dev_W2, *dev_b2;
  double *dev_dW1, *dev_db1, *dev_dW2, *dev_db2;
  
  hipMalloc((void**)&dev_W1, size_W1 * sizeof(double));
  hipMalloc((void**)&dev_b1, size_b1 * sizeof(double));
  hipMalloc((void**)&dev_W2, size_W2 * sizeof(double));
  hipMalloc((void**)&dev_b2, size_b2 * sizeof(double));
  
  hipMalloc((void**)&dev_dW1, size_dW1 * sizeof(double));
  hipMalloc((void**)&dev_db1, size_db1 * sizeof(double));
  hipMalloc((void**)&dev_dW2, size_dW2 * sizeof(double));
  hipMalloc((void**)&dev_db2, size_db2 * sizeof(double));

  // update w1
  hipMemcpy(dev_dW1,  dW1, size_dW1  * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_W1,   W1,  size_W1   * sizeof(double), hipMemcpyHostToDevice);
  update_Wb<<<size_hidden, size_input>>>(dev_dW1, dev_W1, size_input, learn_rate);
  hipMemcpy(W1, dev_W1, size_W1 * sizeof(double), hipMemcpyDeviceToHost);

  // update b1
  hipMemcpy(dev_db1,  db1, size_db1  * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_b1,   b1,  size_b1   * sizeof(double), hipMemcpyHostToDevice);
  update_Wb<<<size_hidden, 1>>>(dev_db1, dev_b1, 1, learn_rate);
  hipMemcpy(b1, dev_b1, size_b1 * sizeof(double), hipMemcpyDeviceToHost);
  
  // update w2
  hipMemcpy(dev_dW2,  dW2, size_dW2  * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_W2,   W2,  size_W2   * sizeof(double), hipMemcpyHostToDevice);
  update_Wb<<<size_output, size_hidden>>>(dev_dW2, dev_W2, size_hidden, learn_rate);
  hipMemcpy(W2, dev_W2, size_W2 * sizeof(double), hipMemcpyDeviceToHost);
  
  // update b2
  hipMemcpy(dev_db2,  db2, size_db2  * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_b2,   b2,  size_b2   * sizeof(double), hipMemcpyHostToDevice);
  update_Wb<<<size_output, 1>>>(dev_db2, dev_b2, 1, learn_rate);
  hipMemcpy(b2, dev_b2, size_b2 * sizeof(double), hipMemcpyDeviceToHost);
  
  /*
  printf("after update**************\n");
    
  
  printf("W1**************\n");
  for (int p = 0; p < 5; p++) {
    for(int q = 0; q < 6; q++) {
      printf("%lf ", W1(p, q));
    }
    printf("\n");
  } 

  printf("b1**************\n");
  for (int p = 0; p < 5; p++) {
    for(int q = 0; q < 1; q++) {
      printf("%lf ", b1(p, q));
    }
    printf("\n");
  } 

  printf("W2**************\n");
  for (int p = 0; p < 2; p++) {
    for(int q = 0; q < 6; q++) {
      printf("%lf ", W2(p, q));
    }
    printf("\n");
  } 

  printf("b2**************\n");
  for (int p = 0; p < 2; p++) {
    for(int q = 0; q < 1; q++) {
      printf("%lf ", b2(p, q));
    }
    printf("\n");
  } 
  */
  
  
  hipFree(dev_W1);
  hipFree(dev_b1);
  hipFree(dev_W2);
  hipFree(dev_b2);

  hipFree(dev_dW1);
  hipFree(dev_db1);
  hipFree(dev_dW2);
  hipFree(dev_db2);

}


void read_X(string data_path, double* array)
{  
  ifstream inFile(data_path);  
  string row;   
  int p;
  p = 0;
  string value;
  while (getline(inFile, row)){  
    stringstream col(row);    
    while (getline(col, value, ',')){
      array[p] = stod(value);      
      p++;
    }   
  }  
}


void read_Y(string data_path, int* array)
{  
  ifstream inFile(data_path);  
  string row;   
  int p;
  p = 0;
  string value;
  while (getline(inFile, row)){  
    stringstream col(row);    
    while (getline(col, value, ',')){
      array[p] = stod(value);      
      p++;
    }   
  }  
}

/* Set the value and reading data */
void read_data()
{

  X_trn = (double *) malloc(size_X_trn * sizeof(double));  // 196*964
  Y_trn = (int *)    malloc(size_Y_trn * sizeof(int));     // 1*964
  X_tst = (double *) malloc(size_X_tst * sizeof(double));  // 196*414
  Y_tst = (int *)    malloc(size_Y_tst * sizeof(int));     // 1*414
  
  
  string X_trn_path = "X_trn.csv"; // Defined the name of cvs file
  string Y_trn_path = "Y_trn.csv";
  string X_tst_path = "X_tst.csv";
  string Y_tst_path = "Y_tst.csv";
        
  read_X(X_trn_path, X_trn); //Execution 
  read_Y(Y_trn_path, Y_trn);  
  read_X(X_tst_path, X_tst);  
  read_Y(Y_tst_path, Y_tst);  

  // Print for test
  /*
  printf("%f\n", X_trn(83, 0));  // 0.125
  printf("%f\n", X_trn(195, 7));  // 0.103515625

  printf("%f\n", X_tst(51, 3));  // 0.092773438
  printf("%f\n", X_tst(55, 9));  // 0.032226563

  printf("%d\n", Y_trn(0, 0));  // 1
  printf("%d\n", Y_trn(0, 6));  // 1
  
  printf("%d\n", Y_tst(0, 2));  // 1
  printf("%d\n", Y_tst(0, 4));  // 0
  */

}

void initialize_Wb() {
  
  W1 = (double *) malloc(size_W1*sizeof(double));   // 20*196
  b1 = (double *) malloc(size_b1*sizeof(double));   // 20*1
  W2 = (double *) malloc(size_W2*sizeof(double));   // 2*20
  b2 = (double *) malloc(size_b2*sizeof(double));   // 2*1
  
  dW1 = (double *) malloc(size_dW1*sizeof(double)); // 20*196
  db1 = (double *) malloc(size_db1*sizeof(double)); // 20*1
  dW2 = (double *) malloc(size_dW2*sizeof(double)); // 2*20
  db2 = (double *) malloc(size_db2*sizeof(double)); // 2*1

  memset (W1,0.5,size_W1);
  memset (b1,0,  size_b1);
  memset (W2,0.5,size_W2);
  memset (b2,0,  size_b2);
  
  memset (dW1,0, size_dW1);
  memset (db1,0, size_db1);
  memset (dW2,0, size_dW2);
  memset (db2,0, size_db2);
  
	default_random_engine e;
	uniform_real_distribution<double> u(-1,1);
 
  for (int i = 0; i < size_W1; i++) {
    W1[i] = u(e);
  }  
  for (int i = 0; i < size_W2; i++) {
    W2[i] = u(e);
  }   
  for (int i = 0; i < size_b1; i++) {
    b1[i] = 0;
  } 
  for (int i = 0; i < size_b2; i++) {
    b2[i] = 0;
  } 
  
}

double accuracy(int* max_index, int* Y, int size_batch) 
{
  
  int i;
  double count = 0;
  for(i = 0; i < size_batch; i++) {
    if(Y(0, i) == max_index(0, i))
      count += 1;
  }  
  return count/double(size_batch);
  
}

double train(double* X_trn, int* Y_trn, int acti_type) {

  forward(X_trn, Y_trn, "train", acti_type);
  backprop(acti_type); // 1 Sigmoid 2 ReLU 
  updateParameter(0.01);
  return cross_entropy_loss(Y_trn, A2, size_train);
  
}

double test(double* X, int* Y, string type, int acti_type) {

  forward(X, Y, type, acti_type);
  if(type == "train")
    return accuracy(max_index, Y, size_train);
  else
    return accuracy(max_index, Y, size_test);
  
}

int main()
{
  
  double loss;
  double acc_trn, acc_tst;
  int e;
  int epochs = 20000;
  int acti_type = 2;
  initialize_Wb();
  read_data();
  float elapsed_time = 0.0;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  for(e = 0; e < 10000; e++) {
    loss = train(X_trn, Y_trn, acti_type);
    // printf("%f\n", loss);
    
    // printf("the %d epoch, the training loss is: %f \n", e, loss);
    acc_trn = test(X_trn, Y_trn, "train", acti_type);
    acc_tst = test(X_tst, Y_tst, "test", acti_type);
    printf("%f\n", acc_trn);
    // printf("the %d epoch, the training accuracy is: %f, the test accuracy is: %f\n", e, acc_trn, acc_tst);
  }
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time, start, stop);
  printf( "Elapsed Time: %.4e msec. \n", elapsed_time );
  
}

